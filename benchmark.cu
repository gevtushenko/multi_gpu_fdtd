#include "hip/hip_runtime.h"
#include "fdtd.cuh"
#include "cuda_benchmark.h"
#include <hip/hip_runtime.h>

__global__ void initialize_fields (
  int n_cells,
  int nx,
  int ny,
  int own_in_process_y_begin,
  float dt,
  float dx,
  float dy,
  float *own_er,
  float *own_hr,
  float *own_mh,
  float *own_hx,
  float *own_hy,
  float *own_ez,
  float *own_dz
)
{
  const unsigned int own_cell_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (own_cell_id < n_cells)
    {
      float er = 1.0;
      float hr = 1.0;

      const int yi = own_in_process_y_begin + own_cell_id / nx;
      const int xi = own_cell_id % nx;

      const float x = static_cast<float> (xi) * dx;
      const float y = static_cast<float> (yi) * dy;

      const float soil_y = static_cast<float> (ny) * dy / 2.2;
      const float object_1_y = soil_y - 8.0;
      const float object_2_y = soil_y - 18.0;
      const float object_1_size = 3.0;
      const float object_2_size = 8.0;

      if (y < soil_y)
        {
          const float middle_x = static_cast<float> (nx) * dx / 2;
          const float object_1_x = middle_x;
          const float object_2_x = middle_x - 20;

          // square
          // if (x > middle_x - object_size / 2.0f && x < middle_x + object_size / 2 && y > object_y - object_size / 2.0 && y < object_y + object_size / 2.0)

          // circle
          if ((x - object_1_x) * (x - object_1_x) + (y - object_1_y) * (y - object_1_y) <= object_1_size * object_1_size)
            er = hr = 200000; /// Relative permeabuliti of Iron
          else if ((x - object_2_x) * (x - object_2_x) + (y - object_2_y) * (y - object_2_y) <= object_2_size * object_2_size)
            er = hr = 200000; /// Relative permeabuliti of Iron
          else
            er = hr = 1.5;
        }

      own_er[own_cell_id] = er;
      own_hr[own_cell_id] = hr;

      own_hx[own_cell_id] = 0.0;
      own_hy[own_cell_id] = 0.0;
      own_ez[own_cell_id] = 0.0;
      own_dz[own_cell_id] = 0.0;

      own_mh[own_cell_id] = C0 * dt / own_hr[own_cell_id];
    }
}

int main ()
{
  hipSetDevice (1);

  /// Grid size equal to block size (1024)
  const int nx = 32;
  const int ny = 32;
  const int n_cells = nx * ny;
  const int n_actual_cells = nx * (2 + ny);

  const float dt = 1e-6;
  const float dx = 1e-1;
  const float dy = 1e-1;

  float *actual_er {};
  float *actual_hr {};
  float *actual_mh {};
  float *actual_hx {};
  float *actual_hy {};
  float *actual_ez {};
  float *actual_dz {};

  hipMalloc (&actual_er, n_actual_cells * sizeof (float));
  hipMalloc (&actual_hr, n_actual_cells * sizeof (float));
  hipMalloc (&actual_mh, n_actual_cells * sizeof (float));
  hipMalloc (&actual_hx, n_actual_cells * sizeof (float));
  hipMalloc (&actual_hy, n_actual_cells * sizeof (float));
  hipMalloc (&actual_ez, n_actual_cells * sizeof (float));
  hipMalloc (&actual_dz, n_actual_cells * sizeof (float));

  float *er = actual_er + nx;
  float *hr = actual_hr + nx;
  float *mh = actual_mh + nx;
  float *hx = actual_hx + nx;
  float *hy = actual_hy + nx;
  float *ez = actual_ez + nx;
  float *dz = actual_dz + nx;

  initialize_fields<<<1, 1024>>> (
    n_actual_cells, nx, ny, 0, dt, dx, dy, er, hr, mh, hx, hy, ez, dz);

  cuda_benchmark::controller controller (1024, 1);

  controller.benchmark ("base h update", [=] __device__ (cuda_benchmark::state &state) {
    const int cell_id = blockIdx.x * blockDim.x + threadIdx.x;

    for (auto _ : state)
      update_h (nx, cell_id, dx, dy, ez, mh, hx, hy);
  });

  controller.benchmark ("shared h update", [=] __device__ (cuda_benchmark::state &state) {
    const int cell_id = threadIdx.x;

    __shared__ float cache[1024 + 32 * 2];

    for (auto _ : state)
      {
        cache[cell_id] = ez[cell_id];
        __syncthreads ();

        update_h (nx, cell_id, dx, dy, cache + 32, mh, hx, hy);
      }
  });

  hipFree (actual_er);
  hipFree (actual_hr);
  hipFree (actual_mh);
  hipFree (actual_hx);
  hipFree (actual_hy);
  hipFree (actual_ez);
  hipFree (actual_dz);

  return 0;
}